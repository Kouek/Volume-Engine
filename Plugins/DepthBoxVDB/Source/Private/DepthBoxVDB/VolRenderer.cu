#include "hip/hip_runtime.h"
#include "VolRenderer.h"

std::unique_ptr<DepthBoxVDB::VolRenderer::IVDBRenderer>
DepthBoxVDB::VolRenderer::IVDBRenderer::Create(const CreateParameters& Params)
{
	if (Params.RHIType == ERHIType::D3D12)
	{
		return std::make_unique<VDBRenderer>(Params);
	}
	return {};
}

struct CUDA_ALIGN VDBStack
{
	DepthBoxVDB::VolData::VDBNode nodes[DepthBoxVDB::VolData::VDBParameters::kMaxLevelNum - 1];
	const DepthBoxVDB::VolData::VDBData& VDBData;
	float	tExits[DepthBoxVDB::VolData::VDBParameters::kMaxLevelNum - 1];
	int32_t Level;

	__host__ __device__ static VDBStack Create(const DepthBoxVDB::VolData::VDBData& VDBData)
	{
		VDBStack Stack = { .VDBData = VDBData, .Level = VDBData.VDBParams.RootLevel + 1 };
		return Stack;
	}
	__host__ __device__ void Push(uint32_t NodeIndex, float tExit)
	{
		--Level;
		nodes[Level - 1] = VDBData.Node(Level, NodeIndex);
		tExits[Level - 1] = tExit;
	}
	__host__ __device__ DepthBoxVDB::VolData::VDBNode& TopNode() { return nodes[Level - 1]; }
	__host__ __device__ float						   TopTExit() { return tExits[Level - 1]; }
	__host__ __device__ void						   Pop() { ++Level; }
	__host__ __device__ bool Empty() { return Level == VDBData.VDBParams.RootLevel + 1; }
	__host__ __device__ bool Full() { return Level == 0; }
};

DepthBoxVDB::VolRenderer::VDBRenderer::VDBRenderer(const CreateParameters& Params)
	: RHIType(Params.RHIType)
{
	int DeviceNum = 0;
	CUDA_CHECK(hipGetDeviceCount(&DeviceNum));
	assert(DeviceNum > 0);

	hipDeviceProp_t Prop;
	CUDA_CHECK(hipGetDeviceProperties(&Prop, 0));
	D3D12NodeMask = Prop.luidDeviceNodeMask;

	CUDA_CHECK(hipStreamCreate(&Stream));
}

DepthBoxVDB::VolRenderer::VDBRenderer::~VDBRenderer()
{
	if (dParams)
	{
		CUDA_CHECK(hipFree(dParams));
	}
}

void DepthBoxVDB::VolRenderer::VDBRenderer::Register(const RegisterParameters& Params)
{
	ID3D12Device*	Device = reinterpret_cast<ID3D12Device*>(Params.Device);
	ID3D12Resource* InSceneDepthTextureNative =
		reinterpret_cast<ID3D12Resource*>(Params.InSceneDepthTexture);
	ID3D12Resource* OutColorTextureNative =
		reinterpret_cast<ID3D12Resource*>(Params.OutColorTexture);

	if (Params.InSceneDepthTexture)
		InSceneDepthTexture = std::make_unique<D3D12::TextureMappedCUDASurface>(
			D3D12NodeMask, Device, InSceneDepthTextureNative);

	OutColorTexture = std::make_unique<D3D12::TextureMappedCUDASurface>(
		D3D12NodeMask, Device, OutColorTextureNative);
	RenderResolution.x = OutColorTexture->TextureDesc.Width;
	RenderResolution.y = OutColorTexture->TextureDesc.Height;

	if (InSceneDepthTexture)
		assert(RenderResolution.x == OutColorTexture->TextureDesc.Width
			&& RenderResolution.y == OutColorTexture->TextureDesc.Height);
}

void DepthBoxVDB::VolRenderer::VDBRenderer::Unregister()
{
	InSceneDepthTexture.reset();
	OutColorTexture.reset();
}

void DepthBoxVDB::VolRenderer::VDBRenderer::SetParameters(const VDBRendererParameters& Params)
{
	bUseDepthBox = Params.bUseDepthBox;
	bUsePreIntegratedTF = Params.bUsePreIntegratedTF;

	if (!dParams)
	{
		CUDA_CHECK(hipMalloc(&dParams, sizeof(VDBRendererParameters)));
	}
	CUDA_CHECK(hipMemcpy(dParams, &Params, sizeof(VDBRendererParameters), hipMemcpyHostToDevice));
}

void DepthBoxVDB::VolRenderer::VDBRenderer::SetTransferFunction(
	const TransferFunctionParameters& Params)
{
	auto Create = [&](const float* Data, const glm::uvec3& Dim) {
		auto Arr = std::make_shared<CUDA::Array>(reinterpret_cast<const float4*>(Data), Dim);

		hipTextureDesc TexDesc{};
		TexDesc.normalizedCoords = 1;
		TexDesc.filterMode = hipFilterModeLinear;
		TexDesc.addressMode[0] = TexDesc.addressMode[1] = TexDesc.addressMode[2] =
			hipAddressModeBorder;
		TexDesc.readMode = hipReadModeElementType;
		return std::make_unique<CUDA::Texture>(Arr, TexDesc);
	};
	TransferFunctionTexture =
		Create(Params.TransferFunctionData, glm::uvec3(Params.Resolution, 1, 1));
	TransferFunctionTexturePreIntegrated = Create(Params.TransferFunctionDataPreIntegrated,
		glm::uvec3(Params.Resolution, Params.Resolution, 1));
}

__device__ static DepthBoxVDB::Ray GenRay(const glm::uvec3& DispatchThreadID,
	const glm::uvec2& RenderResolution, const glm::mat4& InverseProjection,
	const glm::mat3& CameraRotation, const glm::vec3& CameraPosition)
{
	DepthBoxVDB::Ray EyeRay;

	// Map [0, RenderResolution.xy - 1] to (-1, 1)
	glm::vec4 Tmp;
	Tmp.z = RenderResolution.x;
	Tmp.w = RenderResolution.y;
	Tmp.x = (2.f * DispatchThreadID.x + 1.f - Tmp.z) / Tmp.z;
	Tmp.y = (2.f * (RenderResolution.y - 1 - DispatchThreadID.y) + 1.f - Tmp.w) / Tmp.w;

	// Inverseproject
	Tmp.z = 1.f;
	Tmp.w = 1.f;
	Tmp = InverseProjection * Tmp;

	EyeRay.Direction = Tmp;
	EyeRay.Direction = glm::normalize(EyeRay.Direction);
	EyeRay.SceneDepthToPixel = glm::abs(1.f / EyeRay.Direction.z);
	EyeRay.Direction = CameraRotation * EyeRay.Direction;

	EyeRay.Origin = CameraPosition;

	return EyeRay;
}

struct CUDA_ALIGN OnChildPushedParameters
{
	float								 tEnter;
	float								 tExit;
	int32_t								 Level;
	const DepthBoxVDB::VolData::VDBNode& Node;
};

struct CUDA_ALIGN LeafEnteredParameters
{
	float								 tEnter;
	float								 tExit;
	const DepthBoxVDB::VolData::VDBNode& Node;
};

template <typename OnChildPushedType, typename OnSteppedType, typename LeafEnteredType>
struct RayCastVDBCallbacks
{
	OnChildPushedType OnChildPushed;
	OnSteppedType	  OnStepped;
	LeafEnteredType	  LeafEntered;
};

template <typename OnChildPushedType, typename OnSteppedType, typename LeafEnteredType>
__device__ static glm::vec4 RayCastVDB(const DepthBoxVDB::VolData::VDBData& VDBData,
	const DepthBoxVDB::Ray&													EyeRay,
	RayCastVDBCallbacks<OnChildPushedType, OnSteppedType, LeafEnteredType>	Callbacks)
{
	using namespace DepthBoxVDB;

	const VolData::VDBParameters& VDBParams = VDBData.VDBParams;

	Ray::HitShellResult HitShell = EyeRay.HitAABB(glm::vec3(0.f), VDBParams.VoxelPerVolume);
	if (HitShell.tEnter >= HitShell.tExit)
	{
		return glm::vec4(0.f);
	}

	VDBStack Stack = VDBStack::Create(VDBData);
	Stack.Push(0, HitShell.tExit - VolRenderer::Eps);
	HDDA3D Hdda3d = HDDA3D ::Create(HitShell.tEnter + VolRenderer::Eps, EyeRay);
	Hdda3d.Prepare(glm::vec3(0.f), VDBParams.ChildCoverVoxelPerLevels[VDBParams.RootLevel]);

	if constexpr (!std::is_same_v<OnChildPushedType, nullptr_t>)
	{
		OnChildPushedParameters Params{ .tEnter = Hdda3d.tCurr,
			.tExit = Hdda3d.tNext - VolRenderer::Eps,
			.Level = Stack.Level,
			.Node = Stack.TopNode() };
		Callbacks.OnChildPushed(Params);
	}

	while (!Stack.Empty() && [&]() {
#ifdef __CUDA_ARCH__
	#pragma unroll
#endif
		for (uint8_t Axis = 0; Axis < 3; ++Axis)
			if (Hdda3d.ChildCoord[Axis] < 0
				|| Hdda3d.ChildCoord[Axis] >= VDBParams.ChildPerLevels[Stack.Level])
				return false;
		return true;
	}())
	{
		Hdda3d.Next();

		auto& Parent = Stack.TopNode();
		auto  ChildIndex = VDBData.Child(Stack.Level, Hdda3d.ChildCoord, Parent);

		if (ChildIndex != VolData::VDBData::kInvalidChild)
		{
			if (Stack.Level == 1)
			{
				Hdda3d.tCurr += VolRenderer::Eps;

				if constexpr (!std::is_same_v<LeafEnteredType, nullptr_t>)
				{
					LeafEnteredParameters Params{ .tEnter = Hdda3d.tCurr,
						.tExit = Hdda3d.tNext - VolRenderer::Eps,
						.Node = VDBData.Node(0, ChildIndex) };
					if (Callbacks.LeafEntered(Params))
						break;
				}

				Hdda3d.Step();
				if constexpr (!std::is_same_v<OnSteppedType, nullptr_t>)
				{
					Callbacks.OnStepped();
				}
			}
			else
			{
				Stack.Push(ChildIndex, Hdda3d.tNext - VolRenderer::Eps);
				Hdda3d.tCurr += VolRenderer::Eps;
				Hdda3d.Prepare(
					Stack.TopNode().Coord * VDBParams.ChildCoverVoxelPerLevels[Stack.Level + 1],
					VDBParams.ChildCoverVoxelPerLevels[Stack.Level]);

				if constexpr (!std::is_same_v<OnChildPushedType, nullptr_t>)
				{
					OnChildPushedParameters Params{ .tEnter = Hdda3d.tCurr,
						.tExit = Hdda3d.tNext - VolRenderer::Eps,
						.Level = Stack.Level,
						.Node = Stack.TopNode() };
					Callbacks.OnChildPushed(Params);
				}
			}
		}
		else
		{
			Hdda3d.Step();

			if constexpr (!std::is_same_v<OnSteppedType, nullptr_t>)
			{
				Callbacks.OnStepped();
			}
		}

		while (Hdda3d.tCurr >= Stack.TopTExit())
		{
			Stack.Pop();
			if (Stack.Empty())
				break;

			Hdda3d.Prepare(Stack.Level == VDBParams.RootLevel
					? VolData::CoordType(0)
					: Stack.TopNode().Coord * VDBParams.ChildCoverVoxelPerLevels[Stack.Level + 1],
				VDBParams.ChildCoverVoxelPerLevels[Stack.Level]);
		}
	}
}

template <typename VoxelType>
__device__ bool DepthSkip(const glm::vec3& PosInBrick,
	const DepthBoxVDB::VolData::CoordType& MinCoordInAtlasBrick, LeafEnteredParameters& Params,
	const DepthBoxVDB::VolData::VDBData& VDBData, const DepthBoxVDB::Ray& EyeRay)
{
	using namespace DepthBoxVDB;

	const VolData::VDBParameters& VDBParams = VDBData.VDBParams;

	DepthDDA2D DepDda2d;
	if (!DepDda2d.Init(Params.tEnter, VDBParams.ChildPerLevels[0],
			VDBParams.DepthCoordValueInAtlasBrick[0], VDBParams.DepthCoordValueInAtlasBrick[1],
			PosInBrick, EyeRay))
		return false;

	while (true)
	{
		VoxelType Depth = surf3Dread<VoxelType>(VDBData.AtlasSurface,
			sizeof(VoxelType) * (MinCoordInAtlasBrick.x + DepDda2d.CoordInBrick.x),
			MinCoordInAtlasBrick.y + DepDda2d.CoordInBrick.y,
			MinCoordInAtlasBrick.z + DepDda2d.CoordInBrick.z);
		if (Depth <= DepDda2d.Depth + VolRenderer::Eps)
			break;
		if (DepDda2d.tCurr >= Params.tExit)
			return true;

		Params.tEnter = DepDda2d.tCurr;
		DepDda2d.StepNext();
	}
	return false;
}

template <typename VoxelType, bool bUseDepthBox, bool bUsePreIntegratedTF>
__device__ static glm::vec4 RenderScene(hipTextureObject_t TransferFunctionTexture,
	float InputPixelDepth, const DepthBoxVDB::VolRenderer::VDBRendererParameters& RendererParams,
	const DepthBoxVDB::VolData::VDBData& VDBData, const DepthBoxVDB::Ray& EyeRay)
{
	using namespace DepthBoxVDB;

	const VolData::VDBParameters& VDBParams = VDBData.VDBParams;

	glm::vec3 Color(0.f);
	float	  Alpha = 0.f;
	float	  ScalarPrev = -1.f;

	glm::vec3 DeltaPos = RendererParams.Step * EyeRay.Direction;
	int32_t	  StepNum = 1;

	RayCastVDBCallbacks Callbacks = { /* OnChildPushed */ nullptr,
		/* OnStepped */
		[&]() { ScalarPrev = -1.f; },
		/* LeafEntered */
		[&](LeafEnteredParameters& Params) {
			Params.tEnter = RendererParams.Step * glm::ceil(Params.tEnter / RendererParams.Step);
			glm::vec3	MinPosInBrick = glm::vec3(Params.Node.Coord * VDBParams.ChildPerLevels[0]);
			glm::vec3	PosInBrick = EyeRay.Origin + Params.tEnter * EyeRay.Direction - MinPosInBrick;
			VolData::CoordType MinCoordInAtlasBrick =
				Params.Node.CoordInAtlas * VDBParams.VoxelPerAtlasBrick
				+ VDBParams.ApronAndDepthWidth;

			if constexpr (bUseDepthBox)
			{
				if (DepthSkip<VoxelType>(PosInBrick, MinCoordInAtlasBrick, Params, VDBData, EyeRay))
					return false;

				Params.tEnter =
					RendererParams.Step * glm::ceil(Params.tEnter / RendererParams.Step);
				PosInBrick = EyeRay.Origin + Params.tEnter * EyeRay.Direction - MinPosInBrick;
			}

			glm::vec3	MinPosInAtlasBrick(MinCoordInAtlasBrick);
			while (Params.tEnter < Params.tExit && Params.tEnter <= RendererParams.MaxStepDist
				&& StepNum <= RendererParams.MaxStepNum && [&]() {
#ifdef __CUDA_ARCH__
	#pragma unroll
#endif
					   for (uint8_t Axis = 0; Axis < 3; ++Axis)
						   if (PosInBrick[Axis] < 0.f
							   || PosInBrick[Axis] >= VDBParams.ChildPerLevels[0])
							   return false;
					   return true;
				   }())
			{
				if (Params.tEnter >= InputPixelDepth)
					return true;

				glm::vec3 SamplePos = MinPosInAtlasBrick + PosInBrick;
				float	  Scalar =
					tex3D<float>(VDBData.AtlasTexture, SamplePos.x, SamplePos.y, SamplePos.z);
				if (ScalarPrev < 0.f)
					ScalarPrev = Scalar;

				if constexpr (bUsePreIntegratedTF)
				{
					float4 TFColorAlpha =
						tex2D<float4>(TransferFunctionTexture, ScalarPrev, Scalar);
					Color = Color
						+ (1.f - Alpha) * glm::vec3(TFColorAlpha.x, TFColorAlpha.y, TFColorAlpha.z);
					Alpha = Alpha + (1.f - Alpha) * TFColorAlpha.w;

					ScalarPrev = Scalar;
				}
				else
				{
					float4 TFColorAlpha = tex2D<float4>(TransferFunctionTexture, Scalar, 0.f);
					Color = Color
						+ (1.f - Alpha) * TFColorAlpha.w
							* glm::vec3(TFColorAlpha.x, TFColorAlpha.y, TFColorAlpha.z);
					Alpha = Alpha + (1.f - Alpha) * TFColorAlpha.w;
				}

				if (Alpha >= RendererParams.MaxAlpha)
					return true;

				Params.tEnter += RendererParams.Step;
				PosInBrick += DeltaPos;
				++StepNum;
			}

			return false;
		} };
	RayCastVDB(VDBData, EyeRay, Callbacks);

	return glm::vec4(Color, Alpha);
}

__device__ static glm::vec4 RenderAABB(
	int32_t Level, const DepthBoxVDB::VolData::VDBData& VDBData, const DepthBoxVDB::Ray& EyeRay)
{
	using namespace DepthBoxVDB;

	const VolData::VDBParameters& VDBParams = VDBData.VDBParams;

	glm::vec3 Color(0.f);
	float	  Alpha = 0.f;

	RayCastVDBCallbacks Callbacks = {
		/* OnChildPushed */ [&](const OnChildPushedParameters& Params) {
			if (Level != Params.Level)
				return;

			int32_t	  VoxelPerNode = Level == VDBParams.RootLevel
				  ? VDBParams.ChildCoverVoxelPerLevels[Level] * VDBParams.ChildPerLevels[Level]
				  : VDBParams.ChildCoverVoxelPerLevels[Level + 1];
			glm::vec3 PosInBrick = EyeRay.Origin + Params.tEnter * EyeRay.Direction
				- glm::vec3(Params.Node.Coord * VoxelPerNode);

			Color = Color + (1.f - Alpha) * .5f * PosInBrick / float(VoxelPerNode);
			Alpha = Alpha + (1.f - Alpha) * .5f;
		},
		/* OnStepped */ nullptr,
		/* LeafEntered */
		[&](const LeafEnteredParameters& Params) {
			if (Level != 0)
				return false;

			glm::vec3 PosInBrick = EyeRay.Origin + Params.tEnter * EyeRay.Direction
				- glm::vec3(Params.Node.Coord * VDBParams.ChildPerLevels[0]);

			Color = Color + (1.f - Alpha) * .5f * PosInBrick / float(VDBParams.ChildPerLevels[0]);
			Alpha = Alpha + (1.f - Alpha) * .5f;

			return false;
		}
	};
	RayCastVDB(VDBData, EyeRay, Callbacks);

	return glm::vec4(Color, Alpha);
}

template <typename VoxelType>
__device__ static glm::vec4 RenderDepthBox(
	const DepthBoxVDB::VolData::VDBData& VDBData, const DepthBoxVDB::Ray& EyeRay)
{
	using namespace DepthBoxVDB;

	const VolData::VDBParameters& VDBParams = VDBData.VDBParams;

	glm::vec3 Color(0.f);
	float	  Alpha = 0.f;

	RayCastVDBCallbacks Callbacks = { /* OnChildPushed */ nullptr,
		/* OnStepped */ nullptr,
		/* LeafEntered */
		[&](const LeafEnteredParameters& Params) {
			glm::vec3 MinPosInBrick = glm::vec3(Params.Node.Coord * VDBParams.ChildPerLevels[0]);
			glm::vec3	PosInBrick = EyeRay.Origin + Params.tEnter * EyeRay.Direction - MinPosInBrick;
			VolData::CoordType MinCoordInAtlasBrick =
				Params.Node.CoordInAtlas * VDBParams.VoxelPerAtlasBrick
				+ VDBParams.ApronAndDepthWidth;

			Alpha = 1.f;
			DepthDDA2D	DepDda2d;
			if (DepDda2d.Init(Params.tEnter, VDBParams.ChildPerLevels[0],
					VDBParams.DepthCoordValueInAtlasBrick[0],
					VDBParams.DepthCoordValueInAtlasBrick[1], PosInBrick, EyeRay))
			{
				float Depth = surf3Dread<VoxelType>(VDBData.AtlasSurface,
					sizeof(VoxelType) * (MinCoordInAtlasBrick.x + DepDda2d.CoordInBrick.x),
					MinCoordInAtlasBrick.y + DepDda2d.CoordInBrick.y,
					MinCoordInAtlasBrick.z + DepDda2d.CoordInBrick.z);
				Color = glm::vec3(Depth / float(VDBParams.ChildPerLevels[0]));

				// Debug FaceIndex
				// if (Depth == 0 || Depth == 1)
				//	Color.r = Color.g = 1.f;
				// else if (Depth == 2 || Depth == 3)
				//	Color.g = 1.f;
				// else if (Depth == 4 || Depth == 5)
				//	Color.b = 1.f;
			}
			else
			{
				Color.r = 1.f;
			}

			return true; // Break at the first leaf entered
		} };
	RayCastVDB(VDBData, EyeRay, Callbacks);

	return glm::vec4(Color, Alpha);
}

__device__ static glm::vec4 RenderPixelDepth(float InputPixelDepth)
{
	using namespace DepthBoxVDB;

	int32_t Division = glm::ceil(InputPixelDepth / 255.f);
	Division = glm::min(Division, 3);

	float	  Remained = InputPixelDepth;
	glm::vec3 Color(0.f);
	for (int32_t i = 0; i < Division; ++i)
	{
		Color[i] = glm::min(Remained, 255.f);
		Remained -= Color[i];
	}
	Color /= 255.f;

	return glm::vec4(Color, 1.f);
}

void DepthBoxVDB::VolRenderer::VDBRenderer::Render(const RenderParameters& Params)
{
	if (!InSceneDepthTexture || !OutColorTexture)
	{
		std::cerr << "Empty Mapped CUDA Texture/Surface(s)\n";
		return;
	}
	if (!InSceneDepthTexture->IsComplete() || !OutColorTexture->IsComplete())
	{
		std::cerr << "Incomplete Mapped CUDA Texture/Surface(s)\n";
		return;
	}
	if (!TransferFunctionTexture || !TransferFunctionTexturePreIntegrated)
	{
		std::cerr << "Empty CUDA Texture/Surface(s)\n";
		return;
	}
	if (!TransferFunctionTexture->IsComplete()
		|| !TransferFunctionTexturePreIntegrated->IsComplete())
	{
		std::cerr << "Incomplete CUDA Texture/Surface(s)\n";
		return;
	}
	if (!dParams)
	{
		std::cerr << "Empty dParams\n";
		return;
	}

	const VolData::VDBBuilder& Builder = static_cast<const VolData::VDBBuilder&>(Params.Builder);
	const VolData::VDBData*	   dVDBData = Builder.GetDeviceData();
	if (!dVDBData)
	{
		std::cerr << "Empty Device Data\n";
		return;
	}

	const VolData::VDBParameters& VDBParams = Builder.VDBParams;

	switch (VDBParams.VoxelType)
	{
		case VolData::EVoxelType::UInt8:
			if (bUseDepthBox && bUsePreIntegratedTF)
				render<uint8_t, true, true>(Params, dVDBData);
			else if (!bUseDepthBox && bUsePreIntegratedTF)
				render<uint8_t, false, true>(Params, dVDBData);
			else if (bUseDepthBox && !bUsePreIntegratedTF)
				render<uint8_t, true, false>(Params, dVDBData);
			else
				render<uint8_t, false, false>(Params, dVDBData);
			break;
		case VolData::EVoxelType::Float32:
			if (bUseDepthBox && bUsePreIntegratedTF)
				render<float, true, true>(Params, dVDBData);
			else if (!bUseDepthBox && bUsePreIntegratedTF)
				render<float, false, true>(Params, dVDBData);
			else if (bUseDepthBox && !bUsePreIntegratedTF)
				render<float, true, false>(Params, dVDBData);
			else
				render<float, false, false>(Params, dVDBData);
			break;
	}

	CUDA_CHECK(hipStreamSynchronize(Stream));
}

template <typename VoxelType, bool bUseDepthBox, bool bUsePreIntegratedTF>
void DepthBoxVDB::VolRenderer::VDBRenderer::render(
	const RenderParameters& Params, const VolData::VDBData* dVDBData)
{
	auto RenderKernel = [InverseProjection = Params.InverseProjection,
							CameraRotation = Params.CameraRotationToLocal,
							CameraPosition = Params.CameraPositionToLocal,
							RenderResolution = RenderResolution,
							InSceneDepthSurface = InSceneDepthTexture
								? InSceneDepthTexture->SurfaceObject
								: hipSurfaceObject_t(0),
							OutColorSurface = OutColorTexture->SurfaceObject,
							TransferFunctionTexture = bUsePreIntegratedTF
								? TransferFunctionTexturePreIntegrated->Get()
								: TransferFunctionTexture->Get(),
							VDBData = dVDBData,
							RendererParams =
								dParams] __device__(const glm::uvec3& DispatchThreadID) {
		if (DispatchThreadID.x >= RenderResolution.x || DispatchThreadID.y >= RenderResolution.y)
			return;

		Ray EyeRay = GenRay(
			DispatchThreadID, RenderResolution, InverseProjection, CameraRotation, CameraPosition);

		auto GetPixelDepth = [&]() {
			if (InSceneDepthSurface == 0 || !RendererParams->bUseDepthOcclusion)
				return 3.4028234e38f; // std::numeric_limits<float>::max()

			return EyeRay.SceneDepthToPixel
				* surf2Dread<float>(
					InSceneDepthSurface, sizeof(float) * DispatchThreadID.x, DispatchThreadID.y);
		};

		glm::vec4 Color;
		switch (RendererParams->RenderTarget)
		{
			case ERenderTarget::Scene:
			{
				float InputPixelDepth = GetPixelDepth();
				Color = RenderScene<VoxelType, bUseDepthBox, bUsePreIntegratedTF>(
					TransferFunctionTexture, InputPixelDepth, *RendererParams, *VDBData, EyeRay);
			}
			break;
			case ERenderTarget::AABB0:
			case ERenderTarget::AABB1:
			case ERenderTarget::AABB2:
				Color = RenderAABB(static_cast<int32_t>(RendererParams->RenderTarget)
						- static_cast<int32_t>(ERenderTarget::AABB0),
					*VDBData, EyeRay);
				break;
			case ERenderTarget::DepthBox:
				Color = RenderDepthBox<VoxelType>(*VDBData, EyeRay);
				break;
			case ERenderTarget::PixelDepth:
			{
				float InputPixelDepth = GetPixelDepth();
				// Debug tExit
				// Ray::HitShellResult HitShell =
				//	EyeRay.HitAABB(glm::vec3(0.f), VDBData->VDBParams.VoxelPerVolume);
				// if (HitShell.tEnter >= HitShell.tExit)
				//{
				//	Color = glm::vec4(0.f);
				//	break;
				//}
				// InputPixelDepth = HitShell.tExit;

				Color = RenderPixelDepth(InputPixelDepth);
			}
			break;
		}

		Color = glm::clamp(Color * 255.f, 0.f, 255.f);
		uchar4 ColorUCh4{ Color.r, Color.g, Color.b, Color.a };

		surf2Dwrite(
			ColorUCh4, OutColorSurface, sizeof(uchar4) * DispatchThreadID.x, DispatchThreadID.y);
	};

	dim3 ThreadPerBlock(CUDA::ThreadPerBlockX2D, CUDA::ThreadPerBlockY2D, 1);
	dim3 BlockPerGrid((RenderResolution.x + ThreadPerBlock.x - 1) / ThreadPerBlock.x,
		(RenderResolution.y + ThreadPerBlock.y - 1) / ThreadPerBlock.y);
	CUDA::ParallelFor(BlockPerGrid, ThreadPerBlock, RenderKernel, Stream);
}
template void DepthBoxVDB::VolRenderer::VDBRenderer::render<uint8_t, true, true>(
	const RenderParameters& Params, const VolData::VDBData* dVDBData);
template void DepthBoxVDB::VolRenderer::VDBRenderer::render<uint8_t, false, true>(
	const RenderParameters& Params, const VolData::VDBData* dVDBData);
template void DepthBoxVDB::VolRenderer::VDBRenderer::render<uint8_t, true, false>(
	const RenderParameters& Params, const VolData::VDBData* dVDBData);
template void DepthBoxVDB::VolRenderer::VDBRenderer::render<uint8_t, false, false>(
	const RenderParameters& Params, const VolData::VDBData* dVDBData);
template void DepthBoxVDB::VolRenderer::VDBRenderer::render<float, true, true>(
	const RenderParameters& Params, const VolData::VDBData* dVDBData);
template void DepthBoxVDB::VolRenderer::VDBRenderer::render<float, false, true>(
	const RenderParameters& Params, const VolData::VDBData* dVDBData);
template void DepthBoxVDB::VolRenderer::VDBRenderer::render<float, true, false>(
	const RenderParameters& Params, const VolData::VDBData* dVDBData);
template void DepthBoxVDB::VolRenderer::VDBRenderer::render<float, false, false>(
	const RenderParameters& Params, const VolData::VDBData* dVDBData);
